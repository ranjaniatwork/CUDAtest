#include <hip/hip_runtime.h>

__global__ void enhanceKernel(const unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int idx = y * width + x;
    float pixel = input[idx] * 1.25f + 15.0f;
    output[idx] = min(255, static_cast<int>(pixel));
}

extern "C" void launchEnhanceKernel(const unsigned char* d_input, unsigned char* d_output, int width, int height) {
    dim3 blockDim(16, 16);
    dim3 gridDim((width + 15) / 16, (height + 15) / 16);
    enhanceKernel<<<gridDim, blockDim>>>(d_input, d_output, width, height);
    hipDeviceSynchronize();
}