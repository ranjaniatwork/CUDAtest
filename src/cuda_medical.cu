#include <iostream>
#include <string>
#include "host_utils.h"

int main(int argc, char* argv[]) {
    std::string input_dir = "./data/input_images";
    std::string output_dir = "./data/output_images";
    std::string mode = "enhance"; // default

    parseCommandLineArgs(argc, argv, input_dir, output_dir, mode);

    std::vector<Image> images = loadImagesFromDirectory(input_dir);

    for (auto& img : images) {
        unsigned char* d_input;
        unsigned char* d_output;

        allocateDeviceMemory(img.width * img.height, &d_input, &d_output);

        hipMemcpy(d_input, img.data, img.width * img.height, hipMemcpyHostToDevice);

        if (mode == "enhance") {
            launchEnhanceKernel(d_input, d_output, img.width, img.height);
        }

        hipMemcpy(img.data, d_output, img.width * img.height, hipMemcpyDeviceToHost);
        saveImage(output_dir, img.filename, img);

        freeDeviceMemory(d_input, d_output);
    }

    return 0;
}